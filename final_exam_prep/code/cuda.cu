
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include "timer.h"
// #define DEBUG
//nvcc -o dot dot_prod.cu

static int n;
static float dot_parallel;
static float dot_serial;

float Serial_dot_prod(float x[], float y[], int n) {//serial
    float cdot = 0.0;

    for (int i = 0; i < n; i++) {
        cdot += x[i] * y[i];
    }
    return cdot;
}

//GPU Kernel
__global__ void dot_1(float *a, float *b, float *c, int n)//basic parallel, assume thred num = vector size
{
    // TODO: Exercise 1
    //compute each threads' corresponding array index.
    //then, use atomicAdd to add the result to c; you can assume each thread will be responsible for only one element
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    float temp = a[index] * b[index];
    atomicAdd(c, temp);
    printf("%f", temp);


}

//GPU Kernel
__global__ void dot_2a(float *a, float *b, float *c, int n)//block distribution
{
    // TODO: Exercise 2a: block distribution
    int np = gridDim.x*blockDim.x;
    int index =  (blockDim.x * blockIdx.x) + threadIdx.x;
    int work ;
    int step =  n/np;
    // printf("index %d",index);
    if(np > n){
        work = 1;
    }
    
    else if ((n % np) == 0){
        work = step;
        // int x = 300 % 20;
        // printf("index %d",work);
    }

    else {
        work = (step);
        // printf("index %d",index);
        if (index == (np-1)){
            
            work = step + (n%np);
            // printf("%d", work);
        }
    }

    float temp =0.0;
    
    for (int i =0; i < work; i++) {
        temp += a[index *step+i] * b[index*step+i];
        
        //  printf("working %d",ind
    }
}