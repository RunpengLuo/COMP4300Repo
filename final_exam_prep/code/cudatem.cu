#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

static int n;
static float dot_parallel;
static float dot_serial;

float Serial_dot_prod(float x[], float y[], int n) {//serial
    float cdot = 0.0;

    for (int i = 0; i < n; i++) {
        cdot += x[i] * y[i];
    }
    return cdot;
}

//GPU Kernel
__global__ void dot_1(float *a, float *b, float *c, int n)//basic parallel, assume thred num = vector size
{
    // TODO: Exercise 1
    //compute each threads' corresponding array index.
    //then, use atomicAdd to add the result to c; you can assume each thread will be responsible for only one element
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    float temp = a[index] * b[index];
    atomicAdd(c, temp);
    printf("%f", temp);


}

//GPU Kernel
__global__ void dot_2a(float *a, float *b, float *c, int n)//block distribution
{
    // TODO: Exercise 2a: block distribution
    int np = gridDim.x*blockDim.x;
    int index =  (blockDim.x * blockIdx.x) + threadIdx.x;
    int work ;
    int step =  n/np;
    // printf("index %d",index);
    if(np > n){
        work = 1;
    }
    
    else if ((n % np) == 0){
        work = step;
        // int x = 300 % 20;
        // printf("index %d",work);
    }

    else {
        work = (step);
        // printf("index %d",index);
        if (index == (np-1)){
            
            work = step + (n%np);
            // printf("%d", work);
        }
    }

    float temp =0.0;
    
    for (int i =0; i < work; i++) {
        temp += a[index *step+i] * b[index*step+i];
        
        //  printf("working %d",ind
    }
}
__device__ void Comparator(int * x, int * y){
    int acc = *x;
    *x = acc > *y ? *y : acc;
    *y = acc > *y ? acc : *y;
    return;
}
// size = 4
// run on single block, with only 2 thread
// arr is allocated on gpu global memory
// return on arr
__global__ void BM4(int * arr){
    __device__ __shared__ int * arr_b;
    hipMalloc(arr_b, sizeof(int) *4);

    for (int i = 0; i < 4; i++){
        arr_b[i] = arr[i];
    }
    if (threadIdx.x == 0){
        Comparator(arr_b, arr_b+2);
    }else {
        Comparator(arr_b+1, arr_b+3);
    }
    __syncthreads();
    if (threadIdx.x == 0){
        Comparator(arr_b, arr_b+1);
    }else {
        Comparator(arr_b+2, arr_b+3);
    }

    for (int i = 0; i < 4; i++){
        arr[i] = arr_b[i];
    }
    return;
}

__host__ void sort(){
    int * arrb = malloc(sizeof(int) * 4);
    arrb[0] = 10;
    arrb[1] = 20;
    arrb[2] = 9;
    arrb[3] = 5;
    __device__  int * arr;
    hipMalloc(arr, sizeof(int) * 4);
    hipMemcpy(arr, arrb, sizeof(int) * 4, hipMemcpyHostToDevice);

    BM4<<<1,2>>>(arr);

    hipMemcpy(arrb, arr, sizeof(int) * 4, hipMemcpyDeviceToHost);
    printf("after BM4=");
    for (int i = 0; i < 3; i ++){
        printf("%d,", arrb[i]);
    }
    printf("%d\n", arrb[3]);
    hipFree(arr);
    free(arrb);
    return;
}